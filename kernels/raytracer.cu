#include "hip/hip_runtime.h"
#include "raytracer.h"
struct RayStack{
    ray r;
    isect i;
    Vec3f colorC;
    int state;
};

__global__ 
void runRayTracerKernel(Scene_d scene, int depth);

__global__ 
void runRayTracerKernelRec(Scene_d scene, int depth, Light_h hostLight);

__device__ 
Vec3f traceRay(Scene_d* scene, ray& r, int depth);

void RayTracer::run(){
    int blockSize = 32;
    dim3 blockDim(blockSize, blockSize); //A thread block is 32x32 pixels
    dim3 gridDim(deviceScene.imageWidth/blockDim.x, deviceScene.imageHeight/blockDim.y);
    int stackDepth = ( 1 << depth) - 1;
    //runRayTracerKernel<<<gridDim, blockDim, stackDepth*sizeof(RayStack)>>>(deviceScene, depth);
    runRayTracerKernelRec<<<gridDim, blockDim>>>(deviceScene, depth, hostLight);
}

__global__
void runRayTracerKernelRec(Scene_d scene, int depth, Light_h hostLight){

    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = py*scene.imageWidth + px;

    float x = float(px)/float(scene.imageWidth);
    float y = float(py)/float(scene.imageHeight);

    //Get view from the camera
    //perturb
    //x += randx; //in [0,1]
    //y += randy; //in [0,1]
    ray r;
    scene.camera.rayThrough(x, y, r);
    Vec3f colorC;
    Light light(&scene, hostLight);
    scene.light = &light;
    colorC = traceRay(&scene, r, depth);

    scene.image[idx] = colorC;

}

__device__ 
Vec3f traceRay(Scene_d* scene, ray& r, int depth){
    isect i;
    Vec3f colorC;

    // std::default_random_engine generator;
    // std::normal_distribution<float> distribution(0.0,0.01);
    if(scene->intersect(r, i)) {
        // YOUR CODE HERE
        Vec3f q = r.at(i.t);

        // An intersection occurred!  We've got work to do.  For now,
        // this code gets the material for the surface that was intersected,
        // and asks that material to provide a color for the ray.  

        // This is a great place to insert code for recursive ray tracing.
        // Instead of just returning the result of shade(), add some
        // more steps: add in the contributions from reflected and refracted
        // rays.
        const Material& m = i.material;	  
        colorC = m.shade(scene, r, i);
        if(depth <= 0) return colorC;
        if(m.Refl()){
            // std::cout<< "HERE"<< std::endl;

            Vec3f Rdir = -2.0*(r.getDirection()*i.N)*i.N + r.getDirection();
            normalize(Rdir);
            
            ray R(q, Rdir);
            colorC += m.kr % traceRay(scene, R, depth - 1);
        }

        // Now handle the Transmission (Refraction)
        if(m.Trans()){


            Vec3f n = i.N;
            Vec3f rd = r.getDirection();
            Vec3f rcos = n*(-rd*n);
            Vec3f rsin = rcos + rd;
            float etai = 1.0;
            float etat = m.ior;
            Vec3f tcos, tsin;
            float eta;
            if(rd*n < 0){
                eta = etai/etat;
                n = -n;
            } else{
                eta = etat/etai;
            }
            tsin = eta*rsin;
            float TIR = 1 - tsin*tsin;
            if(TIR >= 0){
                tcos = n*sqrt(TIR);
                Vec3f Tdir = tcos + tsin;
                normalize(Tdir);

                ray T(q, Tdir);
                colorC += m.kt % traceRay(scene, T, depth - 1);

            }

        }

    } else {
        // No intersection.  This ray travels to infinity, so we color
        // it according to the background color, which in this (simple) case
        // is just black.
        colorC = Vec3f(0.0, 0.0, 0.0);
    }
    return colorC;

}
__global__ 
void runRayTracerKernel(Scene_d scene, int depth){
    extern __shared__ RayStack rayStack[];
    RayStack* stackPtr = rayStack;
    int curDepth = 0;

    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = py*scene.imageWidth + px;

    float x = float(px)/float(scene.imageWidth);
    float y = float(py)/float(scene.imageHeight);

    //Get view from the camera
    //perturb
    //x += randx; //in [0,1]
    //y += randy; //in [0,1]
    //scene.camera.rayThrough(x, y, stackPtr->r);


    while(true){
        ray& r = stackPtr->r;
        isect& i = stackPtr->i;
        Vec3f& colorC = stackPtr->colorC;
        int& state = stackPtr->state;

        if(state == 0) //Check for intersection
        {
            if(scene.intersect(r, i)){
                Vec3f q = r.at(i.t);
                colorC = i.material.shade(&scene, r, i);
                if(curDepth >= depth){state = 5;} //Exit
                else
                    state = 1;
            }else{
                colorC = Vec3f(0.0,0.0,0.0);
            }
        }
        if(state == 1) //Check for reflection
        {
            if(!i.material.Refl())
                state = 3;
            else{
                Vec3f Rdir = -2.0*(r.getDirection()*i.N)*i.N + r.getDirection();
                normalize(Rdir);

                //Put DRT stuff HERE

                state = 2; //Select next state for my stack frame return
                Vec3f q = r.at(i.t);

                //Push
                stackPtr++;
                curDepth++;

                stackPtr->r = ray(q, Rdir);
                stackPtr->state = 0;
                continue; //Handle the stack push
            }
        }
        if(state == 2) //Post reflection
        {
            colorC += i.material.kr % (stackPtr+1)->colorC;
            state = 3;
        }
        if(state == 3) //Check for refraction
        {
            if(!i.material.Trans())
                state = 5; // Done
            else{
                Vec3f n = i.N;
                Vec3f rd = r.getDirection();
                Vec3f rcos = n*(-rd*n);
                Vec3f rsin = rcos + rd;
                float etai = 1.0;
                float etat = i.material.ior;
                Vec3f tcos, tsin;
                float eta;
                if(rd*n < 0){
                    eta = etai/etat;
                    n = -n;
                } else{
                    eta = etat/etai;
                }
                tsin = eta*rsin;
                float TIR = 1 - tsin*tsin;
                if(TIR >= 0){
                    tcos = n*sqrt(TIR);
                    Vec3f Tdir = tcos + tsin;
                    Vec3f q = r.at(i.t);
                    normalize(Tdir);

                    //Put DRT stuff HERE

                    //Recusive part
                    state = 4;

                    //Push
                    stackPtr++;
                    curDepth++;

                    stackPtr->r = ray(q, Tdir);
                    stackPtr->state = 0;
                    continue; //Handle the stack push
                }
            }
        }
        if(state == 4) //Post refraction
        {

            colorC += i.material.kt % (stackPtr+1)->colorC;
            state = 5;
        }
        // There is no state 5 on purpose
        if(curDepth == 0) //Hit nothing and am at root of stack
            break;
        else{
            stackPtr--; //Pop
            curDepth--;
        }

    }

    scene.image[idx] = rayStack[0].colorC;
}
