#include "hip/hip_runtime.h"
#include "raytracer.h"
#include "debug.h"
#include "mrand.h"
#include "indexing.h"
#include <stdio.h>
#include <iostream>

#define PRINTVEC3(x) printf_DEBUG("(%f,%f,%f)",x.x,x.y,x.z)
struct RayStack{
    ray r;
    isect i;
    Vec3d colorC;
    int state;
};


__global__ 
void runRayTracerKernelRec(Scene_d* scene, int depth);

__device__ 
Vec3d traceRay(Scene_d* scene, ray& r, int depth);
__global__
void initLight(Scene_d* scene, Light_h hostLight, Light* light){
    printf_DEBUG("Adding Light to scene\n");
    *light = Light(scene, hostLight);
    scene->light = light;
}
__global__
void initScene(Scene_d* scene, Scene_d ht){
        scene->numVertices = ht.numVertices;
        scene->imageWidth = ht.imageWidth;
        scene->imageHeight = ht.imageHeight;
        scene->numTriangles = ht.numTriangles;
        scene->numMaterials = ht.numMaterials;

        scene->vertices = ht.vertices;
        scene->normals = ht.normals;
        scene->texcoords = ht.texcoords;

        scene->materials = ht.materials;
        scene->material_ids = ht.material_ids;

        scene->BBoxs = ht.BBoxs; //Per Triangle Bounding Box

        scene->t_indices = ht.t_indices;

        scene->image = ht.image;
        scene->bvh = ht.bvh;

        scene->light = ht.light;
        scene->camera = ht.camera;

        scene->seeds = ht.seeds;
}
void RayTracer::run(){
    int blockSize = 16;
    dim3 blockDim(blockSize, blockSize); //A thread block is 16x16 pixels
    dim3 gridDim(deviceScene.imageWidth/blockDim.x, deviceScene.imageHeight/blockDim.y);

    Scene_d* scene;
    Light* light;
    hipDeviceSynchronize();
    std::cout << "Start of Ray Trace run " << hipGetErrorString(hipGetLastError()) << std::endl;
    hipMalloc(&scene, sizeof(Scene_d));
    hipMalloc(&light, sizeof(Light));
    hipMemcpy(scene, &deviceScene, sizeof(Scene_d), hipMemcpyHostToDevice);
//    initScene<<<1,1>>>(scene, deviceScene);
    std::cout << "Before Light Kernel " << hipGetErrorString(hipGetLastError()) << std::endl;

    initLight<<<1,1>>>(scene, hostLight, light);
    std::cout << "Post Init Light " << hipGetErrorString(hipGetLastError()) << std::endl;
    printf("\nLaunching Ray tracer kernel\n");
    hipDeviceSynchronize();
    size_t stackSize;
//    hipDeviceSetLimit(hipLimitStackSize, 1 << 16);
    std::cout << "Post Set Stack Limit " << hipGetErrorString(hipGetLastError()) << std::endl;
    hipDeviceGetLimit(&stackSize, hipLimitStackSize);
    printf_DEBUG("Stack size is %d\n", stackSize);
    std::cout << "Post Get Stack Limit " << hipGetErrorString(hipGetLastError()) << std::endl;
    
    runRayTracerKernelRec<<<gridDim, blockDim>>>(scene, depth);
    std::cout << "Post Ray Trace " << hipGetErrorString(hipGetLastError()) << std::endl;
    hipDeviceSynchronize();
    printf("\nDone rendering Scene\n");

    hipFree(scene); //Conveniently does not call destructor
    hipFree(light);
}

__global__
void runRayTracerKernelRec(Scene_d* scene, int depth){
    //printf("in kernel\n");

    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = (scene->imageHeight - py - 1)*scene->imageWidth + px;

    if (px >= scene->imageWidth)
        return;
    if (py >= scene->imageHeight)
        return;
    if (idx < 0 || idx >= scene->imageHeight * scene->imageWidth)
        return;
    //           double x = double(px)/double(scene->imageWidth);
    //           double y = double(py)/double(scene->imageHeight);

    // Initialize the PRNG
    initPNRG(scene->seeds);
    //Get view from the camera
    //perturb
    //x += randx; //in [0,1]
    //y += randy; //in [0,1]
    //           ray r;
    //           scene->getCamera()->rayThrough(x, y, r);
    /*    double invWidth = 1.0 / double(scene->imageWidth), invHeight = 1.0 / double(scene->imageHeight);
          double fov = 35, aspectratio = double(scene->imageWidth) / double(scene->imageHeight);
          double angle = tan(M_PI * 0.5 * fov / 180.0f);
          double xx = (2 * ((px + 0.5) * invWidth) - 1)*angle*aspectratio;
          double yy = (1 - 2 * ((py + 0.5) * invHeight)) * angle;
          double focalDistance = 0.0433/(2.0*angle);
    //double focalDistance = 70.0/1000.0;
    double focalPoint = 7;
    //double lenseDistance = 1.0/(1.0/focalDistance - 1.0/focusPoint); //Doesnt matter
    double dofAngle = 2*M_PI*randDouble(scene->seeds);
    double dofRadius = scene->getCamera()->getAperature()*focalDistance * sqrt(randDouble(scene->seeds)) / 2.0;
    Vec3d origin(dofRadius*cos(dofAngle), dofRadius*sin(dofAngle), 0);
    //ray r(Vec3d(0.0,0.0,0.0), Vec3d(xx, yy, -1));
    ray r(origin, Vec3d(xx, yy, 1.0));
    r.d = origin - normalize(r.d)*focalPoint;
    normalize(r.d);
    */ 
    double invWidth = 1.0 / double(scene->imageWidth), invHeight = 1.0 / double(scene->imageHeight);
    double fov = 35, aspectratio = double(scene->imageWidth) / double(scene->imageHeight);
    double focalPoint = 7;
    double angle = tan(M_PI * 0.5 * fov / 180.0f);
    double xx = (2 * ((px + 0.5) * invWidth) - 1)*angle*aspectratio;
    double yy = (1 - 2 * ((py + 0.5) * invHeight)) * angle;
    double focalDistance = 0.0433/(2.0*angle);
    Vec3d colorC;
    int N = 5;
    //double focalDistance = 70.0/1000.0;
    //double lenseDistance = 1.0/(1.0/focalDistance - 1.0/focusPoint); //Doesnt matter
    for(int iter = 0; iter < N; iter++){
        double dofAngle = 2*M_PI*randDouble(scene->seeds);
        double dofRadius = scene->getCamera()->getAperature() * sqrt(randDouble(scene->seeds)) / 2.0;
        Vec3d origin(dofRadius*cos(dofAngle), dofRadius*sin(dofAngle), 0);
        ray r(origin, Vec3d(xx, yy, -1.0));

        r.d = normalize(r.d)*focalPoint - origin;
        //temp remove th drt with following 2 lines
        r.p = Vec3d(0,0,0);
        r.d = Vec3d(xx,yy,-1);
        normalize(r.d);

        printf_DEBUG("RAY %d, p=(%f,%f,%f), d=(%f,%f,%f)\n", idx, r.p.x, r.p.y, r.p.z, r.d.x, r.d.y, r.d.z);
        //printf_DEBUG("Attempting to trace ray\n");
        colorC += traceRay(scene, r, depth);
    }
    colorC /= double(N);
    scene->image[idx] = colorC;

}

__device__ 
Vec3d traceRay(Scene_d* scene, ray& r, int depth){
    isect* i = new isect();
    Vec3d colorC;

    // std::default_random_engine generator;
    // std::normal_distribution<double> distribution(0.0,0.01);
 //   printf("traceRay\n");
    if(scene->intersect(r, *i)) {
        // YOUR CODE HERE
        Vec3d q = r.at(i->t);

        //printf_DEBUG("q=(%f,%f,%f)\n", q.x, q.y, q.z);
        // An intersection occurred!  We've got work to do.  For now,
        // this code gets the material for the surface that was intersected,
        // and asks that material to provide a color for the ray.  

        // This is a great place to insert code for recursive ray tracing.
        // Instead of just returning the result of shade(), add some
        // more steps: add in the contributions from reflected and refracted
        // rays.
        const Material* m = &scene->materials[scene->material_ids[i->object_id]]; //i->material;	  
        colorC = m->shade(scene, r, *i);
        //        printf_DEBUG("colorC=(%f,%f,%f)\n", colorC.x, colorC.y, colorC.z);
        if(depth <= 0){
            delete i;
            return colorC;
        }
        /*
           if(m.Refl()){
        // std::cout<< "HERE"<< std::endl;

        Vec3d Rdir = -2.0*(r.getDirection()*i.N)*i.N + r.getDirection();
        normalize(Rdir);

        ray R(q, Rdir);
        colorC += m.kr % traceRay(scene, R, depth - 1);
        }
        // Now handle the Transmission (Refraction)
        if(m.Trans()){


        Vec3d n = i.N;
        Vec3d rd = r.getDirection();
        Vec3d rcos = n*(-rd*n);
        Vec3d rsin = rcos + rd;
        double etai = 1.0;
        double etat = m.ior;
        Vec3d tcos, tsin;
        double eta;
        if(rd*n < 0){
        eta = etai/etat;
        n = -n;
        } else{
        eta = etat/etai;
        }
        tsin = eta*rsin;
        double TIR = 1 - tsin*tsin;
        if(TIR >= 0){
        tcos = n*sqrt(TIR);
        Vec3d Tdir = tcos + tsin;
        normalize(Tdir);

        ray T(q, Tdir);
        colorC += m.kt % traceRay(scene, T, depth - 1);

        }

        }
        */

    } else {
        // No intersection.  This ray travels to infinity, so we color
        // it according to the background color, which in this (simple) case
        // is just black.
        //colorC = Vec3d(0.0, 0.0, 0.0);
        colorC = Vec3d(0.9, 0.9, 0.9);
    }
    delete i;
    return colorC;

}
