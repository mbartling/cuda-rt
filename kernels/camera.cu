#include "hip/hip_runtime.h"
#include "camera.h"
#include "ray.h"

#define PI 3.14159265359
#define SHOW(x) (cerr << #x << " = " << (x) << "\n")

using namespace std;

__host__ __device__
Camera::Camera(){
    aspectRatio = 1;
    normalizedHeight = 1;

    eye = Vec3d(0.f,0.f,0.f);
    u = Vec3d(1.f,0.f,0.f);
    v = Vec3d(0.f,1.f,0.f);
    look = Vec3d(0.f,0.f,-1.f);
    aperature = 1;
    fov  = 30;
}

__host__ __device__
    void
Camera::rayThrough(double x, double y, ray &r)
    // Ray through normalized window point x,y.  In normalized coordinates
    // the camera's x and y vary both vary from 0 to 1.
{
    x -= 0.5f;
    y -= 0.5f;
    Vec3d dir = look + x*u + y*v;
    normalize(dir);
    r.p = eye;
    r.d = dir;
}

__host__ __device__
    void
Camera::setEye(const Vec3d &eye)
{
    this->eye = eye;
}

__host__ __device__
void 
Camera::setAperature(double a){ aperature = a; }

__host__ __device__
    void
Camera::setLook(double r, double i, double j, double k)
    // Set the direction for the camera to look using a quaternion.  The
    // default camera looks down the neg z axis with the pos y axis as up.
    // We derive the new look direction by rotating the camera by the
    // quaternion rijk.
{
    //set look matrix
    m.x.x = 1.f - 2.f * (i*i + j*j);
    m.x.y = 2.0 * (r*i - j*k);
    m.x.z = 2.f * (j*r + i*k);

    m.y.x = 2.f * (r*i + j*k);
    m.y.y = 1.f - 2.f * (j*j + r*r) ;
    m.y.z = 2.f * (i*j - r*k);

    m.z.x = 2.f * (j*r - i*k);
    m.z.y = 2.f * (i*j + r*k);
    m.z.z = 1.f - 2.f * (i*i + r*r);

    update();
}

__host__ __device__
    void
Camera::setLook(const Vec3d &viewDir, const Vec3d &upDir)
{
    Vec3d z = -viewDir;
    const Vec3d &y = upDir;
    Vec3d x = y ^ z;

    //m = Mat3d(x.x, x.y, x.z, y.x, y.y, y.z, z.x, z.y, z.z).transpose();
    m = Mat3d(x.x, y.x, z.x, x.y, y.y, z.y, x.z, y.z, z.z);
    update();
}

__host__ __device__
    void
Camera::setFOV( double fov )
    // fov - field of view (height) in degrees
{
    fov /= (180.f / PI); // convert to radians
    normalizedHeight = 2.f * tan(fov/2.f);
    update();
}

__host__ __device__
    void
Camera::setAspectRatio(double ar)
    // ar - ratio of width to height
{
    aspectRatio = ar;
    update();
}

__host__ __device__
    void
Camera::update()
{
    u = m * Vec3d(1.f, 0.f, 0.f) * normalizedHeight*aspectRatio;
    v = m * Vec3d(0.f, 1.f, 0.f) * normalizedHeight;
    look = m * Vec3d(0.f, 0.f, -1.f);
}
